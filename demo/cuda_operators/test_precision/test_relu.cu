#include "relu/relu.cuh"

// int main() {
//   constexpr int N = 1024;
//   std::array<float, N> x_host{1};
//   std::array<float, N> y_host{2};
//
//   float *x_cuda = nullptr;
//   float *y_cuda = nullptr;
//   hipMalloc(&x_cuda, N);
//   hipMalloc(&y_cuda, N);
//
//   auto guard = scope_guard([&]() noexcept {
//     hipFree(x_cuda);
//     hipFree(y_cuda);
//   });
//
//   hipMemcpy(x_cuda, x_host.data(), N,
//   hipMemcpyKind::hipMemcpyHostToDevice); relu<<<1, 1>>>(x_cuda, N, y_cuda);
//
//   hipMemcpy(y_host.data(), y_cuda, N,
//   hipMemcpyKind::hipMemcpyDeviceToHost); print_array(y_host);
// }
