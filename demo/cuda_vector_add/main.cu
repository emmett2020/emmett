// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/0_Introduction/vectorAdd

#include <hip/hip_runtime.h>
#include <array>
#include <cstddef>
#include <cstring>

namespace {
__global__ void VecAdd(const float *A, const float *B, float *C, int N) {
  std::size_t i = (blockDim.x * blockIdx.x) + threadIdx.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

} // namespace

int main() {
  constexpr std::size_t N = 1024;
  std::size_t byte_size = N * sizeof(float);

  // Allocate host memory.
  std::array<float, N> host_a{};
  std::array<float, N> host_b{};
  std::array<float, N> host_c{};

  // Initialize host buffer.
  std::memset(host_a.data(), 0, N);
  std::memset(host_b.data(), 0, N);

  // Allocate device memory
  float *device_a = nullptr;
  float *device_b = nullptr;
  float *device_c = nullptr;
  hipMalloc(&device_a, byte_size);
  hipMalloc(&device_b, byte_size);
  hipMalloc(&device_c, byte_size);

  // Copy buffers from host memory to device memory
  hipMemcpy(device_a, host_a.data(), byte_size, hipMemcpyHostToDevice);
  hipMemcpy(device_b, host_b.data(), byte_size, hipMemcpyHostToDevice);

  // Invoke kernel
  constexpr int threads_per_block = 256;
  constexpr int blocks_per_grid =
      (N + threads_per_block - 1) / threads_per_block;
  VecAdd<<<blocks_per_grid, threads_per_block>>>(device_a, device_b, device_c,
                                                 N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(device_c, host_c.data(), byte_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
}
